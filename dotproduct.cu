/*******************************************************************************
 *
 ******************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 512

#define GPU_ERR_CHK(ans) { gpu_assert((ans), __FILE__, __LINE__); }
static void gpu_assert(hipError_t code, const char *file, int line,
        bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code),
                file, line);
        if (abort) {
            exit(code);
        }
    }
}

__global__ void cu_init(unsigned long long seed, hiprandState_t * states_d,
        size_t size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size) {
        hiprand_init(seed, idx, 0, &states_d[idx]);
    }
}

__global__ void cugen_curand_array(hiprandState_t * states_d, int * array_d,
        size_t size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size) {
        int r = hiprand_uniform(&states_d[idx]) * 100;
        array_d[idx] = r;
    }
}

extern "C" void gen_curand_array(unsigned long long seed, int * array,
        size_t size) {
    int blocks = ceil(size / ((float) BLOCK_SIZE));
    dim3 dimgrid (blocks);
    dim3 dimblock (BLOCK_SIZE);
    hiprandState_t * states_d;
    int * array_d;

    GPU_ERR_CHK(hipMalloc((void **) &states_d, size *
                sizeof(hiprandState_t)));
    cu_init<<<dimgrid, dimblock>>>(seed, states_d, size);

    GPU_ERR_CHK(hipMalloc((void **) &array_d, size * sizeof(int)));
    cugen_curand_array<<<dimgrid, dimblock>>>(states_d, array_d, size);

    GPU_ERR_CHK(hipMemcpy(array, array_d, size * sizeof(int),
                hipMemcpyDeviceToHost));

    GPU_ERR_CHK(hipFree(states_d));
    GPU_ERR_CHK(hipFree(array_d));
}

template <unsigned int block_size>
__global__ void cu_dot(int * a_d, int * b_d, int * block_results_d,
        size_t size) {
    extern __shared__ int cache[];

    unsigned int tid = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    cache[tid] = 0;

    if(idx < size) {
        cache[tid] = a_d[idx] * b_d[idx];
    }

    __syncthreads();
    if(block_size >= 512) {
        if(tid < 256) {
            cache[tid] += cache[tid + 256];
        }
        __syncthreads();
    }

    if(block_size >= 256) {
        if(tid < 128) {
            cache[tid] += cache[tid + 128];
        }
        __syncthreads();
    }

    if(block_size >= 128) {
        if(tid < 64) {
            cache[tid] += cache[tid + 64];
        }
        __syncthreads();
    }

    if(tid < 32) {
        if(block_size >= 64) {
            cache[tid] += cache[tid + 32];
        }
        __syncthreads();

        if(block_size >= 32) {
            cache[tid] += cache[tid + 16];
        }
        __syncthreads();

        if(block_size >= 16) {
            cache[tid] += cache[tid + 8];
        }
        __syncthreads();

        if(block_size >= 8) {
            cache[tid] += cache[tid + 4];
        }
        __syncthreads();

        if(block_size >= 4) {
            cache[tid] += cache[tid + 2];
        }
        __syncthreads();

        if(block_size >= 2) {
            cache[tid] += cache[tid + 1];
        }
    }

    __syncthreads();
    if(tid == 0) {
        block_results_d[blockIdx.x] = cache[0];
    }
}

extern "C" void dot_product(long long * result, int * a, int * b, size_t size) {
    int * a_d;
    int * b_d;
    int blocks = 1;
    if(size > BLOCK_SIZE) {
        blocks = ceil(((float)size) / BLOCK_SIZE);
    }
    int cache_size = BLOCK_SIZE * sizeof(int); 
    int * block_results_d;
    int * block_results = (int *) malloc(blocks * sizeof(int));
    dim3 dimgrid (blocks);
    dim3 dimblock (BLOCK_SIZE);

    GPU_ERR_CHK(hipMalloc((void **) &a_d, sizeof(int) * size));
    GPU_ERR_CHK(hipMalloc((void **) &b_d, sizeof(int) * size));
    GPU_ERR_CHK(hipMalloc((void **) &block_results_d, blocks * sizeof(int)));

    GPU_ERR_CHK(hipMemcpy(a_d, a, sizeof(int) * size,
                hipMemcpyHostToDevice));
    GPU_ERR_CHK(hipMemcpy(b_d, b, sizeof(int) * size,
                hipMemcpyHostToDevice));

    //cu_dot<BLOCK_SIZE> <<<dimgrid, dimblock, cache_size>>>(a_d, b_d, block_results_d, size);
    cu_dot<BLOCK_SIZE> <<<blocks, BLOCK_SIZE, cache_size>>>(a_d, b_d, block_results_d, size);

    GPU_ERR_CHK(hipMemcpy(block_results, block_results_d,
                blocks * sizeof(int), hipMemcpyDeviceToHost));

    for(int z = 0; z < blocks; z++) {
        *result += block_results[z];
    }

    //*result = *block_results;

    hipFree(a_d);
    hipFree(b_d);
    hipFree(block_results_d);
}
