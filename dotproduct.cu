/*******************************************************************************
 *
 ******************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 512

#define GPU_ERR_CHK(ans) { gpu_assert((ans), __FILE__, __LINE__); }
static void gpu_assert(hipError_t code, const char *file, int line,
        bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code),
                file, line);
        if (abort) {
            exit(code);
        }
    }
}

__global__ void cu_init(unsigned long long seed, hiprandState_t * states_d,
        size_t size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size) {
        hiprand_init(seed, idx, 0, &states_d[idx]);
    }
}

__global__ void cugen_curand_array(hiprandState_t * states_d, int * array_d,
        size_t size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size) {
        int r = hiprand_uniform(&states_d[idx]) * 100;
        array_d[idx] = r;
    }
}

extern "C" void gen_curand_array(unsigned long long seed, int * array,
        size_t size) {
    int blocks = ceil(size / ((float) BLOCK_SIZE));
    dim3 dimgrid (blocks);
    dim3 dimblock (BLOCK_SIZE);
    hiprandState_t * states_d;
    int * array_d;

    GPU_ERR_CHK(hipMalloc((void **) &states_d, size *
                sizeof(hiprandState_t)));
    cu_init<<<dimgrid, dimblock>>>(seed, states_d, size);

    GPU_ERR_CHK(hipMalloc((void **) &array_d, size * sizeof(int)));
    cugen_curand_array<<<dimgrid, dimblock>>>(states_d, array_d, size);

    GPU_ERR_CHK(hipMemcpy(array, array_d, size * sizeof(int),
                hipMemcpyDeviceToHost));

    GPU_ERR_CHK(hipFree(states_d));
    GPU_ERR_CHK(hipFree(array_d));
}

__global__ void cu_dot(int * a_d, int * b_d, int * block_results_d, size_t size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ int cache[BLOCK_SIZE];
    if (idx < size) {
        cache[threadIdx.x] = a_d[idx] * b_d[idx];
    }else{
        cache[threadIdx.x] = 0;
    }
    __syncthreads();

    if(threadIdx.x == 0) {
        block_results_d[blockIdx.x] = 0;
        for(int z = 0; z < BLOCK_SIZE; z++) {
            block_results_d[blockIdx.x] += cache[z];
        }
    }
}

extern "C" void dot_product(int * result, int * a, int * b, size_t size) {
    int * a_d;
    int * b_d;
    int blocks = ceil(size / ((float) BLOCK_SIZE));
    int * block_results_d;
    int * block_results = (int *) malloc(blocks * sizeof(int));
    dim3 dimgrid (blocks);
    dim3 dimblock (BLOCK_SIZE);

    GPU_ERR_CHK(hipMalloc((void **) &a_d, sizeof(int) * size));
    GPU_ERR_CHK(hipMalloc((void **) &b_d, sizeof(int) * size));
    GPU_ERR_CHK(hipMalloc((void **) &block_results_d, blocks * sizeof(int)));

    GPU_ERR_CHK(hipMemcpy(a_d, a, sizeof(int) * size,
                hipMemcpyHostToDevice));
    GPU_ERR_CHK(hipMemcpy(b_d, b, sizeof(int) * size,
                hipMemcpyHostToDevice));

    cu_dot <<<dimgrid, dimblock>>> (a_d, b_d, block_results_d, size);


    GPU_ERR_CHK(hipMemcpy(block_results, block_results_d,
                blocks * sizeof(int), hipMemcpyDeviceToHost));

    for(int z = 0; z < blocks; z++) {
        *result += block_results[z];
    }

    hipFree(a_d);
    hipFree(b_d);
    hipFree(block_results_d);
}
